
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>
#include <stdlib.h>


unsigned int N = 1;
unsigned int S = 4;
unsigned int D = 3;

// среднее для скользящего окна
__global__ void add(double *inputArr, double *outputArr, int inputSize, int outputSize) {
    int col = threadIdx.x;
    int row = blockIdx.x;

    //printf("%d, %d \n", col,row);

    int result = 0;

    for (int k = row*2; k < row*2+4; k++) {
        for (int m = col*2; m < col*2+4; m++) {
            result += inputArr[k*inputSize + m];
           // if (col == 1 && row == 1) printf("%d, %d: %f, %d \n",row,col, inputArr[k*inputSize + m], k*inputSize+m);

        }
    }

    outputArr[row*(outputSize) + col] = result / 16;
    //printf("%f, \n", result/16);


}

void fillRandArr(double *Arr, int powD, int powS){
    for (int i = 0; i < powD; i++) {
        for (int j = 0; j < powS; j++) {
            int tmp = rand() % 100; // допустим не больше 100
            Arr[i*powS + j] = tmp;
            printf("%d, ", (int)Arr[i*powS + j]);
        }

        printf("\n");
    }

    printf("\n");

}

void formExpandedArr(double *inputArr, double *expandedArr, int powD, int powS){

    // верх лево
    expandedArr[0] = inputArr[0];
    // верх
    for (int j = 1; j < powS + 1; j++) {
        expandedArr[j] = inputArr[j-1];
    }
    // верх право
    expandedArr[powS + 1] = inputArr[powS-1];
    // право
    for (int i = 1; i < powD + 1; i++) {
        expandedArr[i*(powS + 2) + powS + 1] = inputArr[(i-1)*powS + powS-1];
    }
    // низ право
    expandedArr[(powS + 2)*(powD + 1) + powS + 1] = inputArr[powS*(powD-1) + powS-1];
    // низ
    for (int j = 1; j < powS + 1; j++) {
        expandedArr[(powS + 2)*(powD + 1) + j] = inputArr[(powS)*(powD-1) + j-1];
    }
    // низ лево
    expandedArr[(powD + 1)*(powS + 2)] = inputArr[(powD-1)*powS];
    // лево
    for (int i = 1; i < powD + 1; i++) {
        expandedArr[i*(powS + 2)] = inputArr[(i-1)*powS];
    }
    //центр
    for (int i = 1; i < powD + 1; i++) {
        for (int j = 1; j < powS + 1; j++) {
           expandedArr[i*(powS + 2) + j] = inputArr[(i-1)*powS + j-1];
        }
    }

}

void printArr(double *arr, int powD, int powS){
    for (int i = 0; i < powD; i++) {
        for (int j = 0; j < powS; j++) {
            printf("%d, ", (int)arr[i*powS+j]);
        }

        printf("\n");
    }

    printf("\n");
}

void printVerificationArr(double *expandedArr, int powDres, int powSres, int powS){
    double ArrResultCh[powDres*powSres]; // конечный

    for (int i = 0; i < powDres; i++) {
        for (int j = 0; j < powSres; j++) {

        int result = 0;

        for (int k = i*2; k < i*2+4; k++) {
                for (int m = j*2; m < j*2+4; m++) {
                    result += expandedArr[k*(powS + 2) + m];
                }
            }

            ArrResultCh[i*powSres + j] = result / 16;
            printf("%d, ", (int)ArrResultCh[i*powSres + j]);
        }

        printf("\n");
    }
}

int main(void) {
    srand(time(NULL));

    double *dev_i, *dev_o;

    int powD = (int)(pow( 2.0, (double)D ));
    int powS = (int)(pow( 2.0, (double)S ));

    int powDres = (int)(pow( 2.0, (double)(D - 1) ));
    int powSres = (int)(pow( 2.0, (double)(S - 1) ));

    //Выделить память на GPU
    hipMalloc( (void**)&dev_i,
                   (powD + 2) * (powS + 2) * sizeof(double) );
    hipMalloc( (void**)&dev_o,
                   powDres * powSres * sizeof(double) );


    double ArrM[powD*powS]; // начальный массив М
    fillRandArr(ArrM, powD, powS);


    while (N > 0) {


        double ArrMPlus[(powD + 2) * (powS + 2)]; // начальный массив М с добавлением крайних рядов
        formExpandedArr(ArrM, ArrMPlus, powD, powS);
        printArr(ArrMPlus, powD+2, powS+2);


        //Копируем массив ArrMPlus в dev_i
        hipMemcpy( dev_i, ArrMPlus,
                              (powD + 2) * (powS + 2) * sizeof(double),
                              hipMemcpyHostToDevice );

        add<<<powDres, powSres>>>(dev_i, dev_o, powS+2, powSres);
        hipDeviceSynchronize();


        double ArrResult[powDres * powSres]; // конечный
        //Копируем массив с GPU на CPU
        hipMemcpy( ArrResult, dev_o, powDres * powSres * sizeof(double), hipMemcpyDeviceToHost );

        printArr(ArrResult, powDres, powSres);
        printVerificationArr(ArrMPlus, powDres, powSres, powS);


        D--;
        S--;
        powD = powDres;
        powS = powSres;
        powDres = (int)(pow( 2.0, (double)(D - 1) ));
        powSres = (int)(pow( 2.0, (double)(S - 1) ));

        for (int i = 0; i < powD; i++) {
            for (int j = 0; j < powS; j++) {
                ArrM[i*powS + j] = ArrResult[i*powS + j];
            }
        }

        printf("New Array:\n");
        printArr(ArrM, powD, powS);


        N--;
    }

    hipFree( dev_i );
    hipFree( dev_o );

    return 0;
}